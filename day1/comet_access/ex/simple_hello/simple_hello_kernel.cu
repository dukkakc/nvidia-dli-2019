/*
 * Copyright 1993-2010 NVIDIA 
 *     Corporation.  
 *     All rights reserved.
  */

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel( void ) {
}

int main( void ) {
    mykernel<<<1,1>>>();
    printf( "Hello, GPU World!\n" );
    return 0;
}

