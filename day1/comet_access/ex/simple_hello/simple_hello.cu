/*
* simple_hello.cu
* Copyright 1993-2010 NVIDIA Corporation. 
*    All right reserved
*/

#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>
int main( void ) 
{ 
   int deviceCount;
   hipGetDeviceCount( &deviceCount ); 
   printf("Hello, NVIDIA DLI Workshop! You have %d devices\n", deviceCount ); 
   return 0;
}
